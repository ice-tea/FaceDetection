#include "hip/hip_runtime.h"
#ifndef _INTEGRAL_KERNEL_H_
#define _INTEGRAL_KERNEL_H_

__global__ void IntegralKernel(long *pic, int width, int height) {
	
}

void integral_kernel(long * h_i, int width, int height){
	long * d_pic;
	size_t bytes = width * height * sizeof( long );
	// Allocate memory for each vector on GPU
    hipMalloc(&d_pic, bytes);   
 
    // Copy host vectors to device
    hipMemcpy(d_pic, h_i, bytes, hipMemcpyHostToDevice);

    // Setup the execution configuration
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 dimGrid((width-1)/TILE_WIDTH + 1, (height-1)/TILE_WIDTH + 1, 1);

    // Launch the device computation threads!
    IntegralKernel<<<dimGrid, dimBlock>>>(d_pic, width, height);

    // Copy array back to host
    hipMemcpy(h_i, d_pic, bytes, hipMemcpyDeviceToHost); 

    // Free device matrices
    hipFree(d_pic);
}

#endif // #ifndef _INTEGRAL_KERNEL_H_