#include "hip/hip_runtime.h"
#ifndef _WEAK_TRAIN_H_
#define _WEAK_TRAIN_H_

#define TNUM 6987
#define FNUM 882

//__const__ bool V[TNUM] = {false};
//__const__ double W[TNUM] = {0.0};

__global__ void KernelWeakTrain(int *tindex, int testNum, double validweight, int* indexR, bool* goodR, double* errorR,
        bool * V, double * W) {
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int index[FNUM];
    __shared__ bool good[FNUM];
    __shared__ double error[FNUM];
    __shared__ double positive_error[FNUM];
    __shared__ double negative_error[FNUM];

    index[id] = 0;
    good[id] = true;
    error[id] = 1e20;
    positive_error[id] = validweight;
    negative_error[id] = validweight;

    int pos = id*FNUM;
    for(int i=0; i<testNum; ++i){
        if (V[tindex[pos]]){
            positive_error[pos] -= W[tindex[pos]];

            if (positive_error[pos] < error[pos]){
              //best = TestWeakClassifier(feature, feature.values_[itest].value_ + 1, 1, positive_error);
            }
        }
        else{
            positive_error[pos] += W[tindex[pos]];
            negative_error[pos]= 1.0 - positive_error[pos];

            if (negative_error[pos] < error[pos]){
              //best = TestWeakClassifier(feature, feature.values_[itest].value_ - 1, -1, negative_error);
            }
        }
    }
    indexR[id] = index[id];
    goodR[id] = good[id];
    errorR[id] = error[id];
}
void select_best_gpu(int featureNum, int testNum, bool * valids, double * weights, double validweight, int* featureIndex,
    int & index, bool & good, double & error){

    hipMemcpyToSymbol(HIP_SYMBOL(V), valids, TNUM *sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(W), weights, TNUM *sizeof(double));

    int * d_f_i;
    size_t bytes = featureNum  * testNum * sizeof( int );
    // Allocate memory for each vector on GPU
    hipMalloc(&d_f_i, bytes);   
 
    // Copy host vectors to device
    hipMemcpy(d_f_i, featureIndex, bytes, hipMemcpyHostToDevice);

    //constant
    bool * V;
    hipMalloc(&V, TNUM*sizeof(bool));
    hipMemcpy(V, valids, TNUM*sizeof(bool), hipMemcpyHostToDevice);
    double * W;
    hipMalloc(&W, TNUM*sizeof(double));
    hipMemcpy(W, weights, TNUM*sizeof(double), hipMemcpyHostToDevice);


    // Launch the device computation threads!
    int * d_i;
    bool * d_g;
    double * d_e;
    hipMalloc(&d_i, featureNum*sizeof(int));
    hipMalloc(&d_g, featureNum*sizeof(bool));
    hipMalloc(&d_e, featureNum*sizeof(double));
    KernelWeakTrain<<<1, FNUM>>>(d_f_i, testNum, validweight, d_i, d_g, d_e, V, W);

    // Copy array back to host
    int* r_i = (int*)malloc(featureNum*sizeof(int));
    bool* r_g = (bool*)malloc(featureNum*sizeof(bool));
    double* r_e = (double*)malloc(featureNum*sizeof(double));
    hipMemcpy(r_i, d_i, bytes, hipMemcpyDeviceToHost); 
    hipMemcpy(r_g, d_g, bytes, hipMemcpyDeviceToHost); 
    hipMemcpy(r_e, d_e, bytes, hipMemcpyDeviceToHost); 

    index = 1;
    good = true;
    error = validweight;

    // Free device matrices
    hipFree(d_f_i);
}
#endif // #ifndef _WEAK_TRAIN_H_