#ifndef _WEAK_TRAIN_H_
#define _WEAK_TRAIN_H_

#include <iostream>

#define TNUM 50
#define FNUM 882

//__const__ bool V[TNUM] = {false};
//__const__ double W[TNUM] = {0.0};

__global__ void KernelWeakTrain(int *tindex, int testNum, double validweight, int* indexR, bool* goodR, double* errorR,
        bool * V, double * W) {
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int index[FNUM];
    __shared__ bool good[FNUM];
    __shared__ double error[FNUM];
    __shared__ double positive_error[FNUM];
    __shared__ double negative_error[FNUM];

    index[id] = 0;
    good[id] = true;
    error[id] = 2e20;
    positive_error[id] = validweight;
    negative_error[id] = validweight;

    int pos = id*TNUM;
    for(int i=0; i<testNum; ++i){
        if (V[tindex[pos]]){
            positive_error[id] -= W[tindex[pos]];

            if (positive_error[id] < error[id]){
                errorR[id] = positive_error[id];
                goodR[id] = true;
                indexR[id] = i;
              //best = TestWeakClassifier(feature, feature.values_[itest].value_ + 1, 1, positive_error);
            }
        }
        else{
            positive_error[id] += W[tindex[pos]];
            negative_error[id]= 1.0 - positive_error[id];

            if (negative_error[id] < error[id]){
                errorR[id] = negative_error[id];
                goodR[id] = false;
                indexR[id] = i;
              //best = TestWeakClassifier(feature, feature.values_[itest].value_ - 1, -1, negative_error);
            }
        }
        pos++;
    }
    //indexR[id] = index[id];
    //goodR[id] = good[id];
    //errorR[id] = error[id];
}
void select_best_gpu(int featureNum, int testNum, bool * valids, double * weights, double validweight, int* featureIndex,
    int * indexResult, bool * goodResult, double * errorResult){

    //cudaMemcpyToSymbol(V, valids, TNUM *sizeof(bool));
    //cudaMemcpyToSymbol(W, weights, TNUM *sizeof(double));

    int * d_f_i;
    size_t bytes = featureNum  * testNum * sizeof( int );
    // Allocate memory for each vector on GPU
    hipMalloc(&d_f_i, bytes);   
 
    // Copy host vectors to device
    hipMemcpy(d_f_i, featureIndex, bytes, hipMemcpyHostToDevice);

    std::cout << " Feature test index on device is ";
    for(int i=0; i<TNUM; ++i){
        std::cout << d_f_i[i]<< " ";
    }
    std::cout << std::endl;

    //constant
    bool * V;
    hipMalloc(&V, TNUM*sizeof(bool));
    hipMemcpy(V, valids, TNUM*sizeof(bool), hipMemcpyHostToDevice);
    double * W;
    hipMalloc(&W, TNUM*sizeof(double));
    hipMemcpy(W, weights, TNUM*sizeof(double), hipMemcpyHostToDevice);


    // Launch the device computation threads!
    int * d_i;
    bool * d_g;
    double * d_e;
    hipMalloc(&d_i, featureNum*sizeof(int));
    hipMalloc(&d_g, featureNum*sizeof(bool));
    hipMalloc(&d_e, featureNum*sizeof(double));
    KernelWeakTrain<<<1, FNUM>>>(d_f_i, testNum, validweight, d_i, d_g, d_e, V, W);

    // Copy array back to host
    hipMemcpy(indexResult, d_i, bytes, hipMemcpyDeviceToHost); 
    hipMemcpy(goodResult, d_g, bytes, hipMemcpyDeviceToHost); 
    hipMemcpy(errorResult, d_e, bytes, hipMemcpyDeviceToHost);

    // Free device matrices
    hipFree(V);
    hipFree(W);
    hipFree(d_f_i);
    hipFree(d_i);
    hipFree(d_g);
    hipFree(d_e);
}
#endif // #ifndef _WEAK_TRAIN_H_