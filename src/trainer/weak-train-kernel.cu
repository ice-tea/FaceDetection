#ifndef _WEAK_TRAIN_H_
#define _WEAK_TRAIN_H_

#define TNUM 6987
#define FNUM 882
#define THREADNUM 96

__constant__ bool V[TNUM];
__constant__ double W[TNUM];

__global__ void KernelWeakTrain(int featureNum, int testNum, int *tindex, 
    double validweight, int* indexR, bool* goodR, double* errorR ) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if(id >= FNUM)
        return;

    indexR[id] = 0;
    goodR[id] = true;
    errorR[id] = 2e20;

    double positive_error = validweight;
    double negative_error = validweight;

    double local_best = validweight;
    bool loca_good = true;
    int local_index = 0;

    int pos = id*testNum;
    
    for(int i=0; i<testNum; ++i){
        if (V[tindex[pos]]){
            positive_error -= W[tindex[pos]];

            if (positive_error < local_best){
                local_best = positive_error;
                loca_good = true;
                local_index = i;
            }
        }
        else{
            positive_error += W[tindex[pos]];
            negative_error = 1.0 - positive_error;

            if (negative_error < local_best){
                local_best = negative_error;
                loca_good = false;
                local_index = i;
            }
        }
        pos++;
    }
    
    indexR[id] = local_index;
    goodR[id] = loca_good;
    errorR[id] = local_best;
}
void select_best_gpu(int featureNum, int testNum, bool * valids, double * weights, double validweight, int* featureIndex,
    int * indexResult, bool * goodResult, double * errorResult){

    //Constant Memory
    hipMemcpyToSymbol(HIP_SYMBOL(V), valids, testNum *sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(W), weights, testNum *sizeof(double));

    //Global For each Feature
    int * d_f_i;
    hipMalloc(&d_f_i, featureNum  * testNum * sizeof( int ));
    hipMemcpy(d_f_i, featureIndex, featureNum  * testNum * sizeof( int ), hipMemcpyHostToDevice);

    // Global For Result
    int * d_i;
    bool * d_g;
    double * d_e;
    hipMalloc(&d_i, featureNum *sizeof(int));
    hipMalloc(&d_g, featureNum *sizeof(bool));
    hipMalloc(&d_e, featureNum *sizeof(double));

    KernelWeakTrain<<<(featureNum-1)/THREADNUM + 1, THREADNUM>>> (featureNum, testNum, d_f_i, validweight, d_i, d_g, d_e /*,V, W*/);

    // Copy array back to host
    hipMemcpy(indexResult, d_i, featureNum *sizeof(int), hipMemcpyDeviceToHost); 
    hipMemcpy(goodResult, d_g, featureNum *sizeof(bool), hipMemcpyDeviceToHost); 
    hipMemcpy(errorResult, d_e, featureNum *sizeof(double), hipMemcpyDeviceToHost);

    // Free device matrices
    hipFree(d_f_i);
    hipFree(d_i);
    hipFree(d_g);
    hipFree(d_e);
}
#endif // #ifndef _WEAK_TRAIN_H_